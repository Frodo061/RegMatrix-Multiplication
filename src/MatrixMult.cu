
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <mm_malloc.h>
#include <stdlib.h>

using namespace std;
hipEvent_t start, stop;

void startStopWatch () {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
}

void stopStopWatch () {
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	cout << time << " ms." << endl;
}

__global__
void multMat(float *a,float *b, float *c, int N){
    int linha=blockIdx.y*blockDim.y+threadIdx.y; 
    int coluna=blockIdx.x*blockDim.x+threadIdx.x;
    float sum=0;
    if(coluna<N&&linha<N){
        for(int i=0;i<N;i++)sum+=a[linha*N+i]*b[i*N+coluna];
        c[linha*N+coluna]=sum;
    }
}

void stencil(float *a, float *b, float *c, int N){
    float *devA,*devB, *devC;
    int NQ = N*N;
    hipMalloc((void**) &devA, NQ * sizeof(float));
    hipMalloc((void**) &devB, NQ * sizeof(float));
    hipMalloc((void**) &devC, NQ * sizeof(float));
    
    startStopWatch();
	hipMemcpy(devA,a,NQ*sizeof(float),hipMemcpyHostToDevice);	
	hipMemcpy(devB,b,NQ*sizeof(float),hipMemcpyHostToDevice);	
    stopStopWatch();
    dim3 dimGrid(N,N);
    dim3 dimBlock(1,1);
    startStopWatch();
    multMat<<<dimGrid,dimBlock>>>(devA,devB,devC,N);
    stopStopWatch();
    startStopWatch();
    hipMemcpy(c,devC,NQ*sizeof(float),hipMemcpyDeviceToHost);
    stopStopWatch();
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
}

void newMatrices(float **a, float **b, float **c, int N){
    int i;
    int NQ = N*N;
    *a = (float *)_mm_malloc(NQ * sizeof(float), 32);
    *b = (float *)_mm_malloc(NQ * sizeof(float), 32);
    *c = (float *)_mm_malloc(NQ * sizeof(float), 32);
    for (i = 0; i < NQ; i++){
        (*b)[i] = 1;
        (*a)[i] = ((float)rand()) / ((float)RAND_MAX);
    }
}

int main (int argc, char** argv) {
  	int N = atoi(argv[1]);
    srand(0);
	float *a,*b,*c;
    newMatrices(&a,&b,&c,N);
    stencil(a,b,c,N);
	return 0;
}
