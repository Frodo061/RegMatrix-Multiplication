#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <mm_malloc.h>
#include <stdlib.h>

using namespace std;
hipEvent_t start, stop;

void startStopWatch () {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
}

void stopStopWatch () {
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time = 0;
    hipEventElapsedTime(&time, start, stop);
}

__global__
void multMat(float *a,float *b, float *c, int N){
    int linha=blockIdx.y*blockDim.y+threadIdx.y; 
    int coluna=blockIdx.x*blockDim.x+threadIdx.x;
    float sum=0;
    if(coluna<N&&linha<N){
        for(int i=0;i<N;i++)sum+=a[linha*N+i]*b[i*N+coluna];
        c[linha*N+coluna]=sum;
    }
}

int validate(float *c, int n) {
    //all resulting columns should have the same values
    for(unsigned i = 0; i < n*n; i += n) {
        float tmp = c[i];
        for(unsigned j = 0; j < n && result; j++) {
            if(c[i + j] != tmp) return 0;
        }
    }
    return 1;
}

void stencil(float *a, float *b, float *c, int N){
    float *devA,*devB, *devC;
    int NQ = N*N;
    hipMalloc((void**) &devA, NQ * sizeof(float));
    hipMalloc((void**) &devB, NQ * sizeof(float));
    hipMalloc((void**) &devC, NQ * sizeof(float));
    
	hipMemcpy(devA,a,NQ*sizeof(float),hipMemcpyHostToDevice);	
	hipMemcpy(devB,b,NQ*sizeof(float),hipMemcpyHostToDevice);	
    dim3 dimGrid(N,N);
    dim3 dimBlock(1,1);
    startStopWatch();
    multMat<<<dimGrid,dimBlock>>>(devA,devB,devC,N);
    stopStopWatch();
    hipMemcpy(c,devC,NQ*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(devA);
    hipFree(devB);
    if (hipSuccess==hipGetLastError() && validate(c,N)){
    	cout << time << " ms." << endl;
    }
    else{
        cout << "Matrix Multiplication failed: algorithm is incorrect" << endl;
        hipFree(devC);
        exit(-1);
    }
    hipFree(devC);
}

void newMatrices(float **a, float **b, float **c, int N){
    int i;
    int NQ = N*N;
    *a = (float *)_mm_malloc(NQ * sizeof(float), 32);
    *b = (float *)_mm_malloc(NQ * sizeof(float), 32);
    *c = (float *)_mm_malloc(NQ * sizeof(float), 32);
    for (i = 0; i < NQ; i++){
        (*b)[i] = 1;
        (*a)[i] = ((float)rand()) / ((float)RAND_MAX);
    }
}

int main (int argc, char** argv) {
  	int N = atoi(argv[1]);
    srand(0);
	float *a,*b,*c;
    newMatrices(&a,&b,&c,N);
    stencil(a,b,c,N);
    checker(c,N);
	return 0;
}
